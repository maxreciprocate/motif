
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <vector>
#include <string>
#include <mutex>

__global__
void launch(uint32_t *d_table, char* d_source, uint32_t size, uint8_t *d_lut, int8_t *d_output) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  while (tidx < size) {
    char c = d_source[tidx];

    if (c == 0x4e) {
      tidx += stride;
      continue;
    }

    int vx = d_table[d_lut[c] - 1];
    int idx = tidx;

    while (true) {
      int wordidx = d_table[5 * vx + 4];

      if (wordidx != 0)
        d_output[wordidx - 1] = 1;

      idx += 1;
      if (idx > size || vx == 0)
        break;

      c = d_source[idx];
      if (c == 0x4e || c < 0x41)
        break;

      vx = d_table[5 * vx + d_lut[c] - 1];
    }

    tidx += stride;
  }
}

void match(uint32_t *d_table, char* d_source, uint32_t size, uint8_t *d_lut,
           int8_t *d_output, int8_t* output, int64_t output_size, std::string& source) 
{
  hipMemcpy(d_source, source.data(), source.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_output, output, output_size, hipMemcpyHostToDevice);

  launch<<<8000, 1024>>>(d_table, d_source, source.size(), d_lut, d_output);
  
  hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
}
