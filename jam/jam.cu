#include "hip/hip_runtime.h"
#include <stdint.h>
#include <algorithm>

#include <string>
#include <vector>

#include "jam.h"

#define ARRAY_WIDTH_MAX        ((1 << 16) - 1)
#define ARRAY_WIDTH_MAX_SHIFT  16

#define noteError(msg) \
  { noteErrorM((msg), __FILE__, __LINE__); }

inline void noteErrorM(hipError_t code, const char* file, int line) {
  if (code != hipSuccess) fprintf(stderr, "(cuda): %s %s %d\n", hipGetErrorString(code), file, line);
}

texture<uint32_t, hipTextureType1D> t_table;
texture<uint8_t, hipTextureType1D> t_translation;

__device__ void setMarker(int index, hipSurfaceObject_t resultSurfObj) {
  int col = index % ARRAY_WIDTH_MAX;
  int row = index / ARRAY_WIDTH_MAX;
  //if (col > 65530) printf("col: %d, row: %d\n", col, row);
  surf2Dwrite<uint8_t>(0x31, resultSurfObj, col, row);
}

__global__ void launch(char* d_source, uint32_t size, hipSurfaceObject_t resultSurfObj) {

  //surf2Dwrite<uint8_t>(0x31, resultSurfObj, 65534 * 2, 8);
  uint32_t stride = gridDim.x * blockDim.x;

  for (uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x; tidx < size; tidx += stride) {
    uint32_t vx = 0;

    for (uint32_t idx = tidx; idx < size; ++idx) {
      uint8_t c = d_source[idx];

      if (c == 0x4e) break;

      uint32_t offset = tex1Dfetch(t_translation, c - 0x40);
      vx = tex1Dfetch(t_table, 5 * vx + offset - 1);

      if (vx == 0) break;

      uint32_t wordidx = tex1Dfetch(t_table, 5 * vx + 4);

      if (wordidx != 0) {
        //d_output[wordidx - 1] = 0x31;
        setMarker(wordidx - 1, resultSurfObj);
        //surf1Dwrite<uint8_t>(0x31, resultSurfObj, wordidx - 1);
      }
    }
  }
}

void setup(std::vector<uint32_t>& table) {
  uint8_t* d_translation;

  noteError(hipMalloc((void**)&d_translation, Lut.size()));
  noteError(hipMemcpy(d_translation, &Lut, Lut.size(), hipMemcpyHostToDevice));
  noteError(hipBindTexture(0, t_translation, d_translation));

  uint32_t* d_table;

  noteError(hipMalloc((void**)&d_table, table.size() * sizeof(uint32_t)));
  noteError(hipMemcpy(d_table, table.data(), table.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
  noteError(hipBindTexture(0, t_table, d_table));
}

void match(char* d_source, std::string& source, uint8_t* d_output, std::vector<uint8_t>& output, float* time) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  dim3 dimGrid(64000);
  dim3 dimBlock(1024);

  int index = 655360;
  printf("test div: %d, res: %d\n", index >> 16, index & ARRAY_WIDTH_MAX);
  hipMemcpy(d_source, source.data(), source.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_output, output.data(), output.size(), hipMemcpyHostToDevice);

  // Allocate CUDA arrays in device memory
  hipChannelFormatDesc doubleDesc = hipCreateChannelDesc<uint8_t>();
  hipArray* cuResultArray;

  //printf("ARRAY_WIDTH_MAX: %d/n", ARRAY_WIDTH_MAX);
  //printf("output.size(): %lu\n\n", output.size());

  const size_t resultWidth = ARRAY_WIDTH_MAX;
  const size_t resultHeight = (output.size() >> 16) + ((output.size() & ARRAY_WIDTH_MAX) && 1);

  noteError(hipMallocArray(&cuResultArray, &doubleDesc, resultWidth, resultHeight,
                  hipArraySurfaceLoadStore));
  int val;
  hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface2D, 0);
  //printf("cudaDevAttrMaxSurface1D: %d\n", val);

  // Specify surface
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;

  // Create the surface objects
  resDesc.res.array.array = cuResultArray;
  hipSurfaceObject_t resultSurfObj = 0;
  hipCreateSurfaceObject(&resultSurfObj, &resDesc);

  hipEventRecord(start, 0);
  launch<<<dimGrid, dimBlock>>>(d_source, source.size(), resultSurfObj);

  //printf("test: %d/n", ((int) output.size()) % ARRAY_WIDTH_MAX);
  //hipMemcpy(output.data(), d_output, output.size(), hipMemcpyDeviceToHost);
  //hipMemcpy(output.data(), cuResultArray, output.size(), hipMemcpyDeviceToHost);
  //printf("resultHeight: %d\n", resultHeight);
  noteError(hipMemcpy2DFromArray(output.data(), resultWidth, cuResultArray, 0, 0, resultWidth, resultHeight - 1, hipMemcpyDeviceToHost));
  noteError(hipMemcpy2DFromArray(output.data() + (resultWidth * (resultHeight - 1)), resultWidth, cuResultArray, 0, resultHeight - 1, (output.size() % ARRAY_WIDTH_MAX) + 1, 1, hipMemcpyDeviceToHost));

std::replace(output.begin(), output.end(), 0, 0x30);
  //for (int i = 0; i < 20; ++i) {
  //  printf("%d", output.data()[i]);
  //}
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(time, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  hipFreeArray(cuResultArray);
}
