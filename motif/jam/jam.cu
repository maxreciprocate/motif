#include "hip/hip_runtime.h"
#include <iostream>

#include <string>
#include <vector>

#include "jam.h"

texture<uint32_t, hipTextureType1D> t_table;
texture<uint8_t, hipTextureType1D> t_translation;
hipChannelFormatDesc uint8Desc = hipCreateChannelDesc<uint8_t>();
hipChannelFormatDesc uint32Desc = hipCreateChannelDesc<uint32_t>();

#define N_CODE    5

__device__ void decode_3_char(char encoded_char, uint8_t data[3]) {
  
  uint8_t countN = encoded_char & 0b11;
    
   if (countN == 0) {
    // decode all 3 chars by 2 bits
    encoded_char >>= 2;
    data[0] = encoded_char & 0b11;

    encoded_char >>= 2;
    data[1] = encoded_char & 0b11;

    encoded_char >>= 2;
    data[2] = encoded_char & 0b11;
  } else if (countN == 1) {
    // in this case we also need to get index of N 
    encoded_char >>= 2;
    uint8_t index = encoded_char & 0b11;
    data[index] = N_CODE;
    
    // decode 2 "not N" chars
    for (uint8_t i = 0; i < 3; ++i) {
        if (i == index) continue;
        encoded_char >>= 2;
        data[i] = encoded_char & 0b11;
    }
  } else if (countN == 2) {
    // in these case ("N_N", since we replace "NN" by "N") decode only middle char 
    encoded_char >>= 2;
    data[0] = N_CODE;
    data[1] = encoded_char & 0b11;
    data[2] = N_CODE;
  } else { // countN == 3
    // will be created in future version of the algorithm, when we will analyze 'N'
  }
}

__device__ void p_c(char a) {
    for (int i = 0; i < 8; i++) {
      printf("%d", !!((a << i) & 0x80));
    }
    printf("\n");
}


__host__ void p_c_h(char a) {
    for (int i = 0; i < 8; i++) {
      printf("%d", !!((a << i) & 0x80)); 
    }
    printf("\n");
}

__global__ void launch(char* d_source, uint32_t size, int8_t* d_output) {
  uint32_t stride = gridDim.x * blockDim.x;

  uint8_t data[3];

  for (uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x; tidx < size; tidx += stride) {
    uint32_t vx = 0;

    
    for (uint32_t idx = tidx; idx < size; ++idx) {
      char encoded_char = d_source[idx / 3];
      // p_c(encoded_char);
      decode_3_char(encoded_char, data);

      //printf("data: %d, %d, %d\n", data[0], data[1], data[2]);
      
      uint8_t data_index = idx % 3;

      //printf("tidx: %d idx: %d data_index: %d\n", tidx, idx, data_index);      
        uint8_t offset = data[data_index];

        if (offset == N_CODE) break;

        vx = tex1Dfetch(t_table, vx * 5 + offset);

        //printf("tidx: %d vx: %d\n", tidx, vx);
        if (vx == 0) break;

        uint32_t wordidx = tex1Dfetch(t_table, vx * 5 + 4);

        //printf("tidx: %d wordidx: %d\n", tidx, wordidx);
        
        if (wordidx != 0) d_output[wordidx - 1] = 1;
      
      if (data_index == 2) continue;
      if (++idx == size) break;

        offset = data[data_index + 1];

        if (offset == N_CODE) break;

        vx = tex1Dfetch(t_table, vx * 5 + offset);

        //printf("tidx: %d vx: %d\n", tidx, vx);
        if (vx == 0) break;

        wordidx = tex1Dfetch(t_table, vx * 5 + 4);

        //printf("wordidx: %d\n", wordidx);

        if (wordidx != 0) d_output[wordidx - 1] = 1;
      

      //printf("tidx: %d data_index: %d before break\n", tidx, data_index);

      if (data_index == 1) continue;
      if (++idx == size) break;

      //printf("tidx: %d data_index: %d after break\n", tidx, data_index);

        offset = data[data_index + 2];

        if (offset == N_CODE) break;

        vx = tex1Dfetch(t_table, vx * 5 + offset);

        //printf("tidx: %d vx: %d\n", tidx, vx);
        if (vx == 0) break;

        wordidx = tex1Dfetch(t_table, vx * 5 + 4);

        //printf("wordidx: %d\n", wordidx);

        if (wordidx != 0) d_output[wordidx - 1] = 1;
      
    }
  }
}

void setup(uint32_t*& d_table, std::vector<uint32_t>& table) {
  uint8_t* d_translation;

  noteError(hipMalloc((void**)&d_translation, Lut.size()));
  noteError(hipMemcpy(d_translation, &Lut, Lut.size(), hipMemcpyHostToDevice));
  noteError(hipBindTexture(0, t_translation, d_translation, uint8Desc, Lut.size()));

  noteError(hipMalloc((void**)&d_table, table.size() * sizeof(uint32_t)));
  noteError(hipMemcpy(d_table, table.data(), table.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

  //for (int row = 0; row < (table.size() / 5); ++row) {
  //  for (int col = 0; col < 5; ++col) {
  //    printf("%d ", table[row * 5 + col]);
  //  }
  //  printf("\n");
  //}


  noteError(hipBindTexture(0, t_table, d_table, uint32Desc, table.size() * sizeof(uint32_t)));
}

void match(char* d_source, encodedGenomeData& source, int8_t* d_output, int8_t* output, int64_t output_size) {
  //dim3 dimGrid(1);
  //dim3 dimBlock(source.data.size());
  dim3 dimGrid(std::max(source.data.size() >> 11, static_cast<size_t>(32768)));
  dim3 dimBlock(1024);


  //std::cout << source.data.size() << std::endl;
  //p_c_h(source.data[0]);
  //std::cout << source.real_size << std::endl;
  noteError(hipMemcpy(d_source, source.data.data(), source.data.size(), hipMemcpyHostToDevice));
  noteError(hipMemcpy(d_output, output, output_size, hipMemcpyHostToDevice));

  launch<<<dimGrid, dimBlock>>>(d_source, source.real_size, d_output);

  noteError(hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost));
}

void clear_table(uint32_t* d_table) {
  noteError(hipUnbindTexture(t_table));
  noteError(hipFree(d_table));
}
