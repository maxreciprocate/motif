#include "hip/hip_runtime.h"
#include <iostream>

#include <string>
#include <vector>
#include <chrono>
#include <atomic>

#include "jam.h"

texture<uint32_t, hipTextureType1D> t_table;
texture<uint8_t, hipTextureType1D> t_translation;
hipChannelFormatDesc uint8Desc = hipCreateChannelDesc<uint8_t>();
hipChannelFormatDesc uint32Desc = hipCreateChannelDesc<uint32_t>();

#define N_CODE    5

__device__ void decode_3_char(char encoded_char, volatile uint8_t* data) {
  
  uint8_t countN = encoded_char & 0b11;
    
   if (countN == 0) {
    // decode all 3 chars by 2 bits
    encoded_char >>= 2;
    data[0] = encoded_char & 0b11;

    encoded_char >>= 2;
    data[1] = encoded_char & 0b11;

    encoded_char >>= 2;
    data[2] = encoded_char & 0b11;
  } else if (countN == 1) {
    // in this case we also need to get index of N 
    encoded_char >>= 2;
    uint8_t index = encoded_char & 0b11;
    data[index] = N_CODE;
    
    // decode 2 "not N" chars
    for (uint8_t i = 0; i < 3; ++i) {
        if (i == index) continue;
        encoded_char >>= 2;
        data[i] = encoded_char & 0b11;
    }
  } else if (countN == 2) {
    // in these case ("N_N", since we replace "NN" by "N") decode only middle char 
    encoded_char >>= 2;
    data[0] = N_CODE;
    data[1] = encoded_char & 0b11;
    data[2] = N_CODE;
  } else { // countN == 3
    // will be created in future version of the algorithm, when we will analyze 'N'
  }

  
}

#define NUM_CHARS_IN_BYTE  3
#define BLOCK_DIM          1024

__global__ void launch(char* d_source, uint32_t size, int8_t* d_output) {
  __shared__ uint8_t s_source[BLOCK_DIM * NUM_CHARS_IN_BYTE];

  uint32_t stride = gridDim.x * blockDim.x;

  for (size_t tidx = blockIdx.x * blockDim.x + threadIdx.x; tidx < size; tidx += stride) {
    uint32_t bidx = tidx - threadIdx.x;
    uint32_t vx = 0;

    //if (threadIdx.x < 580) {
      decode_3_char(
        d_source[(bidx / 3) + threadIdx.x],
        &s_source[threadIdx.x * 3]
      );
    //}

    __syncthreads();

    uint16_t s_source_index = threadIdx.x + (bidx % 3);
    for (uint32_t global_index = tidx; global_index < size; ++global_index) {

      uint8_t offset = s_source[s_source_index++];

      if (offset == N_CODE) break;

      vx = tex1Dfetch(t_table, vx * 5 + offset);

      if (vx == 0) break;

      uint32_t wordidx = tex1Dfetch(t_table, vx * 5 + 4);

      if (wordidx != 0) d_output[wordidx - 1] = 1;

    }
  
    __syncthreads();
  }
}

void setup(uint32_t*& d_table, std::vector<uint32_t>& table) {
  uint8_t* d_translation;

  noteError(hipMalloc((void**)&d_translation, Lut.size()));
  noteError(hipMemcpy(d_translation, &Lut, Lut.size(), hipMemcpyHostToDevice));
  noteError(hipBindTexture(0, t_translation, d_translation, uint8Desc, Lut.size()));

  noteError(hipMalloc((void**)&d_table, table.size() * sizeof(uint32_t)));
  noteError(hipMemcpy(d_table, table.data(), table.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

  noteError(hipBindTexture(0, t_table, d_table, uint32Desc, table.size() * sizeof(uint32_t)));
}

void match(char* d_source, encodedGenomeData& source, int8_t* d_output, int8_t* output, int64_t output_size) {
  dim3 dimGrid(std::max(source.data.size() >> 11, static_cast<size_t>(32768)));
  dim3 dimBlock(1024);

  noteError(hipMemcpy(d_source, source.data.data(), source.data.size(), hipMemcpyHostToDevice));
  noteError(hipMemcpy(d_output, output, output_size, hipMemcpyHostToDevice));

  launch<<<dimGrid, dimBlock>>>(d_source, source.real_size, d_output);

  noteError(hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost));
}

void clear_table(uint32_t* d_table) {
  noteError(hipUnbindTexture(t_table));
  noteError(hipFree(d_table));
}
